#include <hip/hip_runtime.h>
#include <core/core.hpp>

using namespace std;
using namespace cv;

typedef unsigned char uchar;

extern hipArray *image1; extern hipArray *image2;
extern hipArray *cost_arr;
extern int width, height; extern int ndisp;

extern hipTextureObject_t texImg1;
extern hipTextureObject_t texImg2;
extern hipTextureObject_t texBin;
extern hipTextureObject_t Tcost;

extern "C" void init_texture(uchar *img1, uchar *img2)
{
    hipMemcpyToArray(image1, 0, 0, img1, width*height*sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpyToArray(image2, 0, 0, img2, width*height*sizeof(uchar), hipMemcpyHostToDevice);

    struct hipResourceDesc resDesc1, resDesc2;
    memset(&resDesc1, 0, sizeof(resDesc1));
    memset(&resDesc2, 0, sizeof(resDesc2));
    resDesc1.resType = hipResourceTypeArray;
    resDesc1.res.array.array = image1;
    resDesc2.resType = hipResourceTypeArray;
    resDesc2.res.array.array = image2;

    struct hipTextureDesc texDesc1, texDesc2;
    memset(&texDesc1, 0, sizeof(texDesc1));
    memset(&texDesc2, 0, sizeof(texDesc2));
    texDesc1.addressMode[0] = hipAddressModeClamp;
    texDesc1.addressMode[1] = hipAddressModeClamp;
    texDesc1.filterMode = hipFilterModePoint;
    texDesc1.readMode = hipReadModeElementType;
    texDesc2.addressMode[0] = hipAddressModeClamp;
    texDesc2.addressMode[1] = hipAddressModeClamp;
    texDesc2.filterMode = hipFilterModePoint;
    texDesc2.readMode = hipReadModeElementType;

    hipCreateTextureObject(&texImg1, &resDesc1, &texDesc1, NULL);
    hipCreateTextureObject(&texImg2, &resDesc2, &texDesc2, NULL);
}

extern "C" void upload_cost(short *h_cost)
{
    hipMemcpyToArray(cost_arr, 0, 0, h_cost, width*ndisp*height*sizeof(short), hipMemcpyHostToDevice);

    struct hipResourceDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.resType = hipResourceTypeArray;
    texDesc.res.array.array = cost_arr;

    struct hipTextureDesc Desc;
    memset(&Desc, 0, sizeof(Desc));
    Desc.addressMode[0] = hipAddressModeClamp;
    Desc.addressMode[1] = hipAddressModeClamp;
    Desc.filterMode = hipFilterModePoint;
    Desc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&Tcost, &texDesc, &Desc, NULL);
}
