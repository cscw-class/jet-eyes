#include <hip/hip_runtime.h>
#include <core/core.hpp>

using namespace cv;
typedef unsigned char uchar;

//Global externs
extern int width;
extern int height;
extern int ndisp;

short* texture1;
short* texture2;
short* textureE3L3;
short* textureL3E3;
short* textureE3E3;

short* convoluted;
short* validpLUT;
short* h_validpLUT;
short* cost;
short* disp; short* disp2;
int* Espace; int* Espace2;
uchar* binary;
uchar *h_binary;
short *h_cost;
Mat mapx1, mapx2, mapy1, mapy2, roi, D2Dmap, ground;
hipArray *image1;
hipArray *image2;
hipArray *bin_tex_arr;
hipArray *cost_arr;


extern "C" void Init_cuda()
{
    hipDeviceReset();
    hipFree(0);

    //mallocs
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipMallocArray(&image1, &channelDesc, width, height);
    hipMallocArray(&image2, &channelDesc, width, height);
    hipChannelFormatDesc texDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipMallocArray(&cost_arr, &texDesc, width*ndisp, height);
    hipMallocArray(&bin_tex_arr, &channelDesc, width, height);
    hipMalloc(&texture1, width*height*sizeof(short));
    hipMalloc(&texture2, width*height*sizeof(short));
    hipMalloc(&textureE3L3, width*height*sizeof(short));
    hipMalloc(&textureL3E3, width*height*sizeof(short));
    hipMalloc(&textureE3E3, width*height*sizeof(short));

    hipMalloc(&convoluted, width*height*sizeof(short));
    hipMalloc(&binary, width*height*sizeof(uchar));

    h_binary = (uchar*)malloc(width*height*sizeof(uchar));
    h_validpLUT = (short*)malloc(width*height*2*sizeof(short));
    h_cost = (short*)malloc(width*ndisp*height*sizeof(short));

    hipMalloc(&validpLUT, width*height*2*sizeof(short));
    hipMalloc(&cost, width*height*ndisp*sizeof(short));
    hipMalloc(&Espace, width*height*ndisp*sizeof(int));
    hipMalloc(&Espace2, width*height*ndisp*sizeof(int));
    hipMalloc(&disp, width*height*sizeof(short));
    hipMalloc(&disp2, width*height*sizeof(short));
}
